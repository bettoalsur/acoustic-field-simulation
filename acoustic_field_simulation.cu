#include "hip/hip_runtime.h"
#include <iostream>
#include "nvidiabook/common/book.h"
#include "timer.h"
#include <math.h> //para operaciones matematicas
#include <fstream>
#include <time.h>


//Parametros de la solucion
#define NTHREADS (512) // numero de hilos por bloque


//Parametros del problema
#define c (343.00)  // velocidad de propagacion en el aire
#define f (40000.00)  //frecuencia de excitacion
#define lambda (c/f)  //longitud de onda
#define omega (2*3.1416*f)  //frecuenca angular


__global__ void campoAcustico(float *d_y ,float *d_z ,float *d_PR, float *d_PI ,float xdis ,float ydis ,float dA,float nel)
{
	int i = threadIdx.x + NTHREADS*blockIdx.x;

	if (i < nel*nel)
	{
		float R = sqrt( xdis*xdis + d_z[i]*d_z[i] + (d_y[i] - ydis)*(d_y[i] - ydis) );
		d_PR[i] += (dA/R)*cos(omega*R/c);
		d_PI[i] += (dA/R)*sin(omega*R/c);
	}
}

__global__ void resultante(float *d_PR, float *d_PI, float *d_PA, float nel)
{
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	if (i < nel*nel)
	{
		d_PA[i] = sqrt( d_PR[i]*d_PR[i] + d_PI[i]*d_PI[i] );
	}
}


int main()
{
	//Campo de muestra
	float ymuestra = 0.040;
	float zmuestra = 0.100;
	int pps = 12;
	int nel;
	if (zmuestra > ymuestra)
	{	nel = (pps/lambda)*zmuestra;	}
	else
	{	nel = (pps/lambda)*ymuestra;	}

	if (nel%2!=0)
	{
		nel+=1;
	}

	//Disco emisor
	float r = 0.025;
	int neldis = ((2*pps*r)/lambda);
	if (neldis%2!=0)
	{
		neldis+=1;
	}

	//Campo && Disco (vacios)
	size_t sizeCampo = nel*nel*sizeof(float);
	size_t sizeDisco = neldis*neldis*sizeof(float);

	float* y = (float*)malloc(sizeCampo);
	float* z = (float*)malloc(sizeCampo);
	float* xdis = (float*)malloc(sizeDisco);
	float* ydis = (float*)malloc(sizeDisco);

	//Llenar los vectores 
	for (int cont1 = 0; cont1 < nel ; cont1++)
	{
		for (int cont2 = 0; cont2 < nel ; cont2++)
		{
			y[cont1+nel*cont2]=cont2*(ymuestra/(nel-1)); //se llena saltando
			z[cont1*nel+cont2]=cont2*(zmuestra/(nel-1)); //se llena en orden
		}
	}

	float dxy = (2*r/(neldis-1));
	float dA = dxy*dxy;
	for (int cont1 = 0; cont1 < neldis ; cont1++)
	{
		for (int cont2 = 0; cont2 < neldis ; cont2++)
		{
			xdis[cont1+neldis*cont2]=-r+cont2*dxy; //se llena saltando
			ydis[cont1*neldis+cont2]=-r+cont2*dxy; //se llena en orden
		}
	}

	//Vector de Presion (vacio) HOST
	float* P = (float*)malloc(sizeCampo);

	
	////// Variables en la GPU //////

	float* d_y; hipMalloc(&d_y,sizeCampo);		//ancho del campo
	float* d_z; hipMalloc(&d_z,sizeCampo);		//largo del campo
	float* d_PR; hipMalloc(&d_PR,sizeCampo);		//Presion REAL en el campo
	float* d_PI; hipMalloc(&d_PI,sizeCampo);		//Presion IMAG en el campo
	float* d_PA; hipMalloc(&d_PA,sizeCampo);		//Presion IMAG en el campo


	// Comienza medicion de TIEMPO
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);


	////// Copia de informacion del Host a la GPU //////

	hipMemcpy(d_y,y,sizeCampo,hipMemcpyHostToDevice);
	hipMemcpy(d_z,z,sizeCampo,hipMemcpyHostToDevice);


	////// Resuleve en la GPU (Kernel) //////

	for (int rr = 0 ; rr < neldis*neldis ; rr++)
	{
		if ( sqrt(xdis[rr]*xdis[rr] + ydis[rr]*ydis[rr]) <= r )
		{
			campoAcustico<<<(nel*nel+(NTHREADS-1))/NTHREADS,NTHREADS>>>(d_y , d_z , d_PR , d_PI , xdis[rr] , ydis[rr] , dA, nel);
		}
	}

	resultante<<<(nel*nel+(NTHREADS-1))/NTHREADS,NTHREADS>>>(d_PR , d_PI, d_PA, nel);


	////// Copia los resultados de la GPU al HOST //////
	hipMemcpy(P,d_PA,sizeCampo,hipMemcpyDeviceToHost);


	// Termina de medir el TIEMPO
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float tempo;
	hipEventElapsedTime(&tempo, start , stop);
	tempo/=1000;
	

	////// Libero la memoria de la GPU //////
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_PR);
	hipFree(d_PI);
	hipFree(d_PA);

	// Mensajes para la consola
	std::cout << "Numero total de elementos: " << nel*nel + neldis*neldis << std::endl;
	std::cout << "Campo: " << nel*nel << " && Disco: " << neldis*neldis << std::endl;
	std::cout << "Tempo CUDA: " << tempo << " segundos" << std::endl;

	//Creacion de los archivos de texto

	std::ofstream file1;
	file1.open ("C:/Users/lemos/proyectoGC/datos.txt");
	for (int contxx = 0 ; contxx < nel*nel ; contxx++)
	{	file1 << y[contxx] << "	" << z[contxx] << "	" << P[contxx] << "\n";  }
	file1.close();

	std::ofstream file2;
	file2.open ("C:/Users/lemos/proyectoGC/tiempo.txt");
	file2 << tempo; 
	file2.close();

	std::ofstream file3;
	file3.open ("C:/Users/lemos/proyectoGC/parametros.txt");
	file3 << c << "	" << f << "	" << ymuestra << "	" << zmuestra << "	" << pps << "	" << r; 
	file3.close();

	///////////////////////////////
	////// Calculo en la CPU //////

	float RR;
	float* Preal = (float*)malloc(sizeCampo);
	float* Pimag = (float*)malloc(sizeCampo);
	float* Pabs = (float*)malloc(sizeCampo);

	double cputime = getTimeStamp();		// Comienza contar tiempo

	for (int rr = 0 ; rr < neldis*neldis ; rr++)
	{
		if ( sqrt(xdis[rr]*xdis[rr] + ydis[rr]*ydis[rr]) <= r )
		{
			for (int ii = 0 ; ii < nel*nel ; ii++)
			{
				RR = sqrt( xdis[rr]*xdis[rr] + z[ii]*z[ii] + (y[ii] - ydis[rr])*(y[ii] - ydis[rr]) );
				Preal[ii] += (dA/RR)*cos(omega*RR/c);
				Pimag[ii] += (dA/RR)*sin(omega*RR/c);
			}
		}
	}

	for (int ii = 0 ; ii < nel*nel ; ii++)
	{
		Pabs[ii] = sqrt( Preal[ii]*Preal[ii] + Pimag[ii]*Pimag[ii] );
	}

	cputime = getTimeStamp() - cputime;		// Termina de contar tiempo
	cputime/=1000;

	std::cout << "Tempo CPU: " << cputime << " segundos" << std::endl;

	//Creacion de los archivos de texto

	std::ofstream file4;
	file4.open ("C:/Users/lemos/proyectoGC/datosCPU.txt");
	for (int contxx = 0 ; contxx < nel*nel ; contxx++)
	{	file4 << y[contxx] << "	" << z[contxx] << "	" << Pabs[contxx] << "\n";  }
	file4.close();

	std::ofstream file5;
	file5.open ("C:/Users/lemos/proyectoGC/tiempoCPU.txt");
	file5 << cputime; 
	file5.close();

	std::cin.get();
	return 0;
}